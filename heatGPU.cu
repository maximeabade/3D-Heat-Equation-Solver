#include "hip/hip_runtime.h"
#include "heatGPU.h"

__global__ void heatGPU(const float * uIn, float * uOut, const dim3 n, const float dt)
{
    
    float hx = 1./n.x;
    float hy = 1./n.y;
    float hz = 1./n.z;

    int i = blockIdx.x * blockDim.x + threadIdx.x + 1;
    int j = blockIdx.y * blockDim.y + threadIdx.y + 1;
    int k = blockIdx.z * blockDim.z + threadIdx.z + 1;

    if (i < n.x - 1 && j < n.y - 1 && k < n.z - 1)
    {
        uOut[i + j * n.x + k * n.x * n.y] = uIn[i + j * n.x + k * n.x * n.y] + dt * (
            (uIn[(i-1) + j * n.x + k * n.x * n.y] + 
             uIn[(i+1) + j * n.x + k * n.x * n.y] ) / (hx * hx) +
            (uIn[i + (j-1) * n.x + k * n.x * n.y] + 
             uIn[i + (j+1) * n.x + k * n.x * n.y] ) / (hy * hy) +
            (uIn[i + j * n.x + (k-1) * n.x * n.y] + 
             uIn[i + j * n.x + (k+1) * n.x * n.y] ) / (hz * hz) 
            -uIn[i + j * n.x + k * n.x * n.y] * 2.0 / (hx *hx)
            -uIn[i + j * n.x + k * n.x * n.y] * 2.0 / (hy *hy)
            -uIn[i + j * n.x + k * n.x * n.y] * 2.0 / (hz *hz));
    }
}
