#include "hip/hip_runtime.h"
#include "heatCPU.h"
#include <stdio.h>

void heatCPU(const float *uIn, float * uOut, const dim3 n, const float dt)
{
    float hx = 1./n.x;
    float hy = 1./n.y;
    float hz = 1./n.z;

    for(int i=1; i < n.x - 1; ++i)
      for(int j=1; j < n.y - 1; ++j)
        for(int k=1; k < n.z - 1; ++k)
          getElem(uOut, n, i, j, k) = getElem(uIn, n, i, j, k) + dt * (
                                      (getElem(uIn, n, i-1, j, k) + 
                                       getElem(uIn, n, i+1, j, k) ) / (hx * hx) +
                                      (getElem(uIn, n, i, j-1, k) + 
                                       getElem(uIn, n, i, j+1, k) ) / (hy * hy) +
                                      (getElem(uIn, n, i, j, k-1) + 
                                       getElem(uIn, n, i, j, k+1) ) / (hz * hz) 
                                      -getElem(uIn, n, i, j, k) * 2.0 / (hx *hx)
                                      -getElem(uIn, n, i, j, k) * 2.0 / (hy *hy)
                                      -getElem(uIn, n, i, j, k) * 2.0 / (hz *hz));

}

void putValOnBoundary(float * u, const dim3 n, const float val)
{    
  // first boundary
  for(int j = 0; j < n.y ; ++j) {
    for (int k = 0; k < n.z ; ++ k) {
      getElem(u, n, 0, j, k) = val;
      getElem(u, n, n.x-1, j, k) = val;
    }
  }
  // second boundary
  for(int i = 0; i < n.x; ++i) {
    for (int k = 0; k < n.z ; ++k) {
      getElem(u, n, i, 0, k) = val;
      getElem(u, n, i, n.y-1, k) = val;
    }
  }
  // third boundary
  for(int i = 0; i < n.x; ++i) {
    for (int j = 0; j < n.y; ++j) {
      getElem(u, n, i, j, 0) = val;
      getElem(u, n, i, j, n.z-1) = val;
    }
  }
}

float computeError(const float * u1, const float * u2, const dim3 n)
{
  float err = 0;
  for (int i = 1; i < n.x - 1; ++i)
    for (int j = 1; j < n.y - 1; ++j)
      for (int k = 1; k < n.z - 1; ++k)
        err += abs(getElem(u1, n, i, j, k) - getElem(u2, n, i, j, k));
  return err;
}

void printTensor(const float * u, const dim3 n)
{
  for (int i = 1; i < n.x - 1; ++i) {
	for (int j = 1; j < n.y - 1; ++j) {
	  for (int k = 1; k < n.z - 1; ++k) {
		printf("%08.2f ", getElem(u, n, i, j, k));
	  }
	  printf("\n");
	}
	printf("\n\n");
  }
}
