#include <stdio.h>
#include "heatCPU.h"
#include "heatGPU.h"
#include "output.h"
#include "hip/hip_runtime_api.h"

int main(int argc, char * argv[])
{
   dim3 grid, tBlock;
   dim3 n;
   float err;
   float * uIn, 
         * uOut, 
         * uCheck;


   float * uInDevice, 
         * uOutDevice;

   size_t size_tot;

   float dt = 1;

   if (argc < 2)
   {
     n = { 8, 8, 8 };
   }
   else
   {
     unsigned int nfix;
     nfix = atoi(argv[1]);
     n = {nfix, nfix, nfix};
   }

   size_tot = sizeof(float) * n.x * n.y * n.z;

   uIn    = (float *) malloc(size_tot);
   uOut   = (float *) malloc(size_tot);
   uCheck = (float *) malloc(size_tot);

   checkCudaErrors(hipMalloc(&uInDevice, size_tot));
   checkCudaErrors(hipMalloc(&uOutDevice, size_tot));

   putValOnBoundary(uIn, n, 1.0);
   putValOnBoundary(uOut, n, 1.0);
   putValOnBoundary(uCheck, n, 1.0);

   checkCudaErrors(hipMemcpy(uInDevice, uIn, size_tot, hipMemcpyHostToDevice));
  
   tBlock = { 8, 8, 8 };

   grid.x = n.x / tBlock.x + (n.x % tBlock.x ? 1 : 0 ) ;
   grid.y = n.y / tBlock.y + (n.y % tBlock.y ? 1 : 0 ) ;
   grid.z = n.z / tBlock.z + (n.z % tBlock.z ? 1 : 0 ) ;


   printf("n : %d %d %d\n", n.x, n.y, n.z);
   printf("grid : %d %d %d\n", grid.x, grid.y, grid.z);
   printf("block : %d %d %d\n", tBlock.x, tBlock.y, tBlock.z);
   heatGPU<<<grid, tBlock>>>(uInDevice, uOutDevice, n, dt);
   checkCudaErrors(hipGetLastError());
   checkCudaErrors(hipDeviceSynchronize());
   checkCudaErrors(hipMemcpy(uCheck, uOutDevice, size_tot, hipMemcpyDeviceToHost));
   free(uIn);
   free(uOut);
   free(uCheck);

   checkCudaErrors(hipFree(uInDevice));
   checkCudaErrors(hipFree(uOutDevice));
   return 0;
}

